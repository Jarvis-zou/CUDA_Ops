#include "hip/hip_runtime.h"
#include "pch.cuh"

typedef float FLOAT;

/* CUDA kernel function */
__global__ void vec_add(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    /* 1D grid */
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = idx; i < N; i += gridDim.x * blockDim.x){
        z[i] = y[i] + x[i];
        //debug info
        //if(i==500) printf("index500,gpuz=%f,y=%f,x=%f\n",z[i],y[i],x[i]);
    }
}

void vec_add_cpu(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    for (int i = 0; i < N; i++) {
        z[i] = y[i] + x[i];
        // debug info
        //if(i==500) printf("i=500,z=%f\n",z[i]);
    }
}

int main()
{
    int N = 10000;
    int nbytes = N * sizeof(FLOAT);
    const int nstreams = 1;
    int nums_per_stream = N / nstreams;// assert N can be exactly divided by nstream
    int size_per_stream = nums_per_stream * sizeof(FLOAT);

    /* 1D block */
    int bs = 256;

    /* 1D grid */
    int s = ceil((size_per_stream + bs - 1.) / bs);
    dim3 grid(s);

    FLOAT *dx, *hx;
    FLOAT *dy, *hy;
    FLOAT *dz, *hz;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, nbytes);
    hipMalloc((void **)&dz, nbytes);

    /* init time */
    float milliseconds = 0;

    /* Allocate Pinned memory on Host for Device to read*/
    hipHostAlloc(&hx, nbytes, hipHostMallocDefault);
    hipHostAlloc(&hy, nbytes, hipHostMallocDefault);
    hipHostAlloc(&hz, nbytes, hipHostMallocDefault);

    /* init */
    for (int i = 0; i < N; i++) {
        hx[i] = 1.0;
        hy[i] = 1.0;
    }
    hipStream_t streams[nstreams];

    // init cuda stream
    for (int i = 0; i < nstreams; i++) {
        hipStreamCreate(&streams[i]);
        //printf("creating %d th stream\n", i);
    }

    // launch cuda kernels
    for(int i = 0; i < nstreams; i++){
        printf("%d th stream is working \n", i);
        int start_per_stream = i * nums_per_stream;
        printf("size_per_steram=%d, start_per_stream=%d\n",size_per_stream,start_per_stream);

        /* async copy data to GPU */
        hipMemcpyAsync(dx + start_per_stream, hx + start_per_stream,
                        size_per_stream, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(dy + start_per_stream, hy + start_per_stream,
                        size_per_stream, hipMemcpyHostToDevice, streams[i]);

        /* launch GPU kernel */
        vec_add<<<grid, bs, 0, streams[i]>>>(dx + start_per_stream, dy + start_per_stream, dz + start_per_stream, nums_per_stream);

        /* async copy GPU result to CPU */
        hipMemcpyAsync(hz + start_per_stream, dz + start_per_stream, size_per_stream, hipMemcpyDeviceToHost, streams[i]);
    }

    // when using ****Async, here we need use hipDeviceSynchronize to sync host and all streams of device.
    // when only need sync one stream and device, use hipStreamSynchronize, which is light-weight
    hipDeviceSynchronize();

    /* CPU compute */
    FLOAT* hz_cpu_res = (FLOAT *) malloc(nbytes);
    vec_add_cpu(hx, hy, hz_cpu_res, N);

    /* check GPU result with CPU*/
    for (int i = 0; i < N; ++i) {
        if (fabs(hz_cpu_res[i] - hz[i]) > 1e-6) {
            printf("index: %d, cpu: %f, gpu: %f\n", i, hz_cpu_res[i], hz[i]);
            break;
        }
    }
    printf("Result right\n");
    for (int i = 0; i < nstreams; i++) {
        hipStreamDestroy(streams[i]);
        printf("destroying %d th stream\n", i);
    }
    hipFree(dx);
    hipFree(dy);
    hipFree(dz);
    // free pinned memory
    hipHostFree(hx);
    hipHostFree(hy);
    hipHostFree(hz);
    free(hz_cpu_res);

    return 0;
}
