#include "hip/hip_runtime.h"
#include "pch.cuh"
#define N 25600000

__global__ void accumulator_baseline(const int *input, int *output) {
    int sum = 0;
    for (int i = 0; i < N; i++) {
        sum += input[i];
    }
    *output = sum;
}

void accumulator_baseline_cpu(const int *input, int *output) {
    int sum = 0;
    for (int i = 0; i < N; i++) {
        sum += input[i];
    }
    *output = sum;
}

int main() {
    int *hin, *hout, *hout_cpu;
    int *din, *dout;

    // allocate CPU data memory
    hin = (int*)malloc(N * sizeof(int));
    hout = (int*)malloc(sizeof(int));
    hout_cpu = (int*)malloc(sizeof(int));

    // init data
    for (int i = 0; i < N; i++) {
        hin[i] = 1;
    }

    //allocate GPU memory
    hipMalloc((void**)&din, N * sizeof(int));
    hipMalloc((void**)&dout, sizeof(int));

    // copy data to GPU
    hipMemcpy(din, hin, N * sizeof(int), hipMemcpyHostToDevice);

    // call kernel function
    float millisecond;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    accumulator_baseline<<<1, 1>>>(din, dout);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millisecond, start, stop);

    // copy GPU result to CPU
    hipMemcpy(hout, dout, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "[GPU] Sum = " << *hout << std::endl;
    std::cout << "Time Spent: " << millisecond << "ms." << std::endl;


    // call CPU function
    accumulator_baseline_cpu(hin, hout_cpu);
    std::cout << "[CPU] Sum = " << *hout_cpu << std::endl;
    if (*hout_cpu != *hout) std::cout << "Result is Wrong!" << std::endl;

    // free resource
    hipFree(din);
    hipFree(dout);

    free(hin);
    free(hout);
    free(hout_cpu);










}
