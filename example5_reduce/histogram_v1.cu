#include "hip/hip_runtime.h"
#include "pch.cuh"
#include <random>
#define N 25600000

__global__ void histogram_v1(const unsigned int *hist_data, unsigned int *block_data, int data_size) {
    // Using shared memory to store local data, avoid collision in global mem(ep: thread0(block0) and thread0(block1)
    // need to do operations on same mem location in global memory) as possible as we can.
    __shared__ unsigned int shared_mem[256];
    unsigned int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    shared_mem[tid] = 0;
    __syncthreads();

    // if total number of threads is smaller than data size, we let some threads handle multiple data
    for (unsigned int i = global_tid; i < data_size; i += gridDim.x * blockDim.x) {
        atomicAdd(&shared_mem[hist_data[i]], 1);
    }
    __syncthreads();

    atomicAdd(&block_data[global_tid], shared_mem[tid]);
}

__global__ void histogram_block_sum(const unsigned int *block_data, unsigned int *bin_data) {
    unsigned int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    atomicAdd(&bin_data[tid], block_data[global_tid]);
}

bool checkResults(const unsigned int *input, unsigned int *groundTruth, int size) {
    for (int i = 0; i < size; i++){
        if (input[i] != groundTruth[i]) {
            printf("Check Failed, out[i]=%d, gt[i]=%d\n", input[i], groundTruth[i]);
            return false;
        }
    }
    return true;
}

int main() {
    unsigned int *h_hist, *h_bin;
    unsigned int *d_hist, *d_bin, *d_block;

    // get device property and calculate block needed
    hipDeviceProp_t deviceProp{};
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);
    const int blockSize = 256;  // 256 threads/block
    int gridSize = std::min(((N + blockSize - 1) / blockSize), deviceProp.maxGridSize[0]);  // keep gridSize because we are not reallocate data to block we are reallocate threads

    // allocate CPU data memory
    h_hist = (unsigned int*)malloc(N * sizeof(unsigned int));
    h_bin = (unsigned int*)malloc(256 * sizeof(unsigned int));  // we only count numbers from 0 to 255

    // init data
    std::random_device rd;
    std::mt19937 gen(rd());
    std::vector<double> weights(blockSize, 1.0);
    for (int i = 0; i < blockSize / 5; ++i) {
        weights[i] = 5.0;
    }
    std::discrete_distribution<> dist(weights.begin(), weights.end());

    unsigned int *groundTruth = (unsigned int*)calloc(256, sizeof(unsigned int));;
    for (int i = 0; i < N; ++i) {
        h_hist[i] = dist(gen);
        groundTruth[h_hist[i]] += 1;
    }
    for (int i = 0; i < 256; i++) {
        std::cout << i << ": " << groundTruth[i] << std::endl;
    }

    //allocate GPU memory
    hipMalloc((void**)&d_hist, N * sizeof(unsigned int));
    hipMalloc((void**)&d_block, N * sizeof(unsigned int));
    hipMalloc((void**)&d_bin, 256 * sizeof(unsigned int));

    // copy data to GPU
    hipMemcpy(d_hist, h_hist, N * sizeof(unsigned int), hipMemcpyHostToDevice);

    // call kernel function
    dim3 Grid(gridSize);  // number of blocks
    dim3 Block(blockSize);  // number of threads
    float millisecond;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    histogram_v1<<<Grid, Block>>>(d_hist, d_block, N);  // first get result of each block
    histogram_block_sum<<<Grid, Block>>>(d_block,d_bin);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millisecond, start, stop);

    // copy GPU result to CPU
    hipMemcpy(h_bin, d_bin, 256 * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Time Spent: " << millisecond << "ms." << std::endl;

    // Check Result
    checkResults(h_bin, groundTruth, 256);

    // free resource
    hipFree(d_hist);
    hipFree(d_bin);

    free(h_hist);
    free(h_bin);
    free(groundTruth);
}
