#include "hip/hip_runtime.h"
#include "pch.cuh"
#include <random>
#define N 25600000

// This warp-level perform same as baseline, both slower than block-level(v1) method 20%
__device__ int atomicAggInc(unsigned int *ctr) {
    unsigned int active = __activemask();
    int leader = __ffs(active) - 1; // leader thread will handle operations on global mem
    int change = __popc(active);// how many threads are active, which means how many value in this warp > 0
    int lane_mask_lt;
    asm("mov.u32 %0, %%lanemask_lt;" : "=r"(lane_mask_lt));
    unsigned int rank = __popc(active & lane_mask_lt); // same logic as block
    int warp_res;
    if(rank == 0)  // only leader thread do add operation
        warp_res = atomicAdd(ctr, change);  //compute global offset of warp
    warp_res = __shfl_sync(active, warp_res, leader);  // broadcast warp_res of leader thread to every active thread
    return warp_res + rank;
}

__global__ void copy_if_v2(const unsigned int *input, unsigned int *dst, unsigned int *global_NCopy, const unsigned int data_size) {
    // Warp-level optimization, divide task in each block into warp-level, reduce more collision on same global memory space
    int global_tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(global_tid >= data_size)
        return;
    if(input[global_tid] > 0)
        dst[atomicAggInc(global_NCopy)] = input[global_tid];
}


int copy_if_cpu(const unsigned int *input, unsigned int *dst, const unsigned int data_size){
    int index = 0;
    for (auto i = 0; i < data_size; i++) {
        if (input[i] > 0) {
            dst[index++] = input[i];
        }
    }
    return index;  // how many values copied
}

bool checkResults(const unsigned int *dst_cpu, const unsigned int *dst_gpu, const unsigned int data_size) {
    for (unsigned int i = 0 ; i < data_size; i++) {
        if (dst_cpu[i] != dst_gpu[i]) {
            std::cout << "Check Failed: res=" << dst_cpu[i] << " Ground Truth=" << dst_gpu[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    unsigned int *h_in, *h_dst, *h_dst_cpu, *h_NCopy;
    unsigned int *d_in, *d_dst, *d_NCopy;

    // get device property and calculate block needed
    hipDeviceProp_t deviceProp{};
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);
    const int blockSize = 256;  // 256 threads/block
    int gridSize = std::min(((N + blockSize - 1) / blockSize), deviceProp.maxGridSize[0]);

    // allocate CPU data memory
    h_in = (unsigned int*)malloc(N * sizeof(unsigned int));
    h_dst = (unsigned int*)malloc(N * sizeof(unsigned int));
    h_dst_cpu = (unsigned int*)malloc(N * sizeof(unsigned int));
    h_NCopy = (unsigned int*)malloc(sizeof(unsigned int));

    // init data
    *h_NCopy = 0;
    for(int i = 0; i < N; i++){
        // all set to 1
        h_in[i] = 1;
    }

    //allocate GPU memory
    hipMalloc((void**)&d_in, N * sizeof(unsigned int));
    hipMalloc((void**)&d_dst, N * sizeof(unsigned int));
    hipMalloc((void**)&d_NCopy, sizeof(unsigned int));

    // copy data to GPU
    hipMemcpy(d_in, h_in, N * sizeof(unsigned int), hipMemcpyHostToDevice);

    // call kernel function
    dim3 Grid(gridSize);  // number of blocks
    dim3 Block(blockSize);  // number of threads
    float millisecond;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    copy_if_v1<<<Grid, Block>>>(d_in, d_dst, d_NCopy, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millisecond, start, stop);

    // copy GPU result to CPU
    hipMemcpy(h_dst, d_dst, N * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(h_NCopy, d_NCopy, sizeof(unsigned int), hipMemcpyDeviceToHost);
    std::cout << "Time Spent: " << millisecond << "ms." << std::endl;

    // Check Result
    *h_NCopy = copy_if_cpu(h_in, h_dst_cpu, N);
    checkResults(h_dst_cpu, h_dst, N);
    if (*h_NCopy != N) {
        std::cout << "Wrong Result:" << " CPU copied: " << N << " GPU copied: " << *h_NCopy << std::endl;
    }
    else{
        std::cout << "Right Result:" << " CPU copied: " << N << " GPU copied: " << *h_NCopy << std::endl;
    }

    // free resource
    hipFree(d_in);
    hipFree(d_dst);
    hipFree(d_NCopy);

    free(h_in);
    free(h_dst);
    free(h_dst_cpu);
    free(h_NCopy);
}
