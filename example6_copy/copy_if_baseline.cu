#include "hip/hip_runtime.h"
#include "pch.cuh"
#include <random>
#define N 25600000

__global__ void copy_if_baseline(const unsigned int *input, unsigned int *dst, unsigned int *NCopy, const unsigned int data_size) {
    const unsigned int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_tid < data_size && input[global_tid] > 0) {
        dst[atomicAdd(NCopy, 1)] = input[global_tid];
    }
}


int copy_if_cpu(const unsigned int *input, unsigned int *dst, const unsigned int data_size){
    int index = 0;
    for (auto i = 0; i < data_size; i++) {
        if (input[i] > 0) {
            dst[index++] = input[i];
        }
    }
    return index;  // how many values copied
}

bool checkResults(const unsigned int *dst_cpu, const unsigned int *dst_gpu, const unsigned int data_size) {
    for (unsigned int i = 0 ; i < data_size; i++) {
        if (dst_cpu[i] != dst_gpu[i]) {
            std::cout << "Check Failed: res=" << dst_cpu[i] << " Ground Truth=" << dst_gpu[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    unsigned int *h_in, *h_dst, *h_dst_cpu, *h_NCopy;
    unsigned int *d_in, *d_dst, *d_NCopy;

    // get device property and calculate block needed
    hipDeviceProp_t deviceProp{};
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);
    const int blockSize = 256;  // 256 threads/block
    int gridSize = std::min(((N + blockSize - 1) / blockSize), deviceProp.maxGridSize[0]);

    // allocate CPU data memory
    h_in = (unsigned int*)malloc(N * sizeof(unsigned int));
    h_dst = (unsigned int*)malloc(N * sizeof(unsigned int));
    h_dst_cpu = (unsigned int*)malloc(N * sizeof(unsigned int));
    h_NCopy = (unsigned int*)malloc(sizeof(unsigned int));

    // init data
    *h_NCopy = 0;
    for(int i = 0; i < N; i++){
        // all set to 1
        h_in[i] = 1;
    }

    //allocate GPU memory
    hipMalloc((void**)&d_in, N * sizeof(unsigned int));
    hipMalloc((void**)&d_dst, N * sizeof(unsigned int));
    hipMalloc((void**)&d_NCopy, sizeof(unsigned int));

    // copy data to GPU
    hipMemcpy(d_in, h_in, N * sizeof(unsigned int), hipMemcpyHostToDevice);

    // call kernel function
    dim3 Grid(gridSize);  // number of blocks
    dim3 Block(blockSize);  // number of threads
    float millisecond;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    copy_if_baseline<<<Grid, Block>>>(d_in, d_dst, d_NCopy, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millisecond, start, stop);

    // copy GPU result to CPU
    hipMemcpy(h_dst, d_dst, N * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(h_NCopy, d_NCopy, sizeof(unsigned int), hipMemcpyDeviceToHost);
    std::cout << "Time Spent: " << millisecond << "ms." << std::endl;

    // Check Result
    *h_NCopy = copy_if_cpu(h_in, h_dst_cpu, N);
    checkResults(h_dst_cpu, h_dst, N);
    if (*h_NCopy != N) {
        std::cout << "Wrong Result:" << " CPU copied: " << N << " GPU copied: " << *h_NCopy << std::endl;
    }
    else{
        std::cout << "Right Result:" << " CPU copied: " << N << " GPU copied: " << *h_NCopy << std::endl;
    }

    // free resource
    hipFree(d_in);
    hipFree(d_dst);
    hipFree(d_NCopy);

    free(h_in);
    free(h_dst);
    free(h_dst_cpu);
    free(h_NCopy);
}
