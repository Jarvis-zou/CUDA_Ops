#include "hip/hip_runtime.h"
#include "../pch.cuh"

__global__ void sum(float *x)
{
    // 泛指当前block在所有block范围内的id
    int block_id = blockIdx.x;
    // 泛指当前线程在所有block范围内的全局id
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    // 泛指当前线程在其block内的id
    int local_tid = threadIdx.x;
    printf("current block=%d, thread id in current block =%d, global thread id=%d\n", block_id, local_tid, global_tid);
    x[global_tid] += 1;
}

int main(){
    int N = 32;
    int nbytes = N * sizeof(float);  // 计算32个float数据的大小，总计32*4=128

    float *dx, *hx; // device指针和host指针

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);//思考为什么要用二级指针

    /* allocate CPU mem */
    hx = (float*)malloc(nbytes);

    /* init host data */
    printf("hx original: \n");
    for (int i = 0; i < N; i++) {
        hx[i] = i;
        printf("%g\n", hx[i]);
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

    /* launch GPU kernel */
    sum<<<1, N>>>(dx);

    /* copy data from GPU */
    hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);
    printf("hx current: \n");
    for (int i = 0; i < N; i++) {
        printf("%g\n", hx[i]);
    }
    hipFree(dx);
    free(hx);

    return 0;
}
