#include "hip/hip_runtime.h"
#include "../pch.cuh"

__global__ void sum(float *x)
{
    int block_id = blockIdx.x;
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    printf("current block=%d, thread id in current block =%d, global thread id=%d\n", block_id, local_tid, global_tid);
    x[global_tid] += 1;
}

int main(){
    int N = 32;
    int nbytes = N * sizeof(float);

    float *dx, *hx;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);

    /* allocate CPU mem */
    hx = (float*)malloc(nbytes);

    /* init host data */
    printf("hx original: \n");
    for (int i = 0; i < N; i++) {
        hx[i] = i;
        printf("%g\n", hx[i]);
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

    /* launch GPU kernel */
    sum<<<1, N>>>(dx);

    /* copy data from GPU */
    hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);
    printf("hx current: \n");
    for (int i = 0; i < N; i++) {
        printf("%g\n", hx[i]);
    }
    hipFree(dx);
    free(hx);

    return 0;
}
