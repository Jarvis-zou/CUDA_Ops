#include "hip/hip_runtime.h"
#include "pch.cuh"
#define N 256000000

template<typename T>
struct BiasMaskScaleAddFunctor {
    const T *bias;
    const int biasSize;
    const uint8_t *mask;
    const T scale;
    const T *add;

    // Init members
    BiasMaskScaleAddFunctor(const float* bias, const int biasSize, const uint8_t *mask, const float scale, const float *add)
            : bias(bias), biasSize(biasSize), mask(mask), scale(scale), add(add) {}

    __device__ T compute(T *x, unsigned int idx) {
//        return __fadd_rn(__fmul_rn(__fmul_rn(__fadd_rn(x[idx], bias[idx % biasSize]), static_cast<float>(mask[idx])), scale), add[idx]);
        return (x[idx] + bias[idx % biasSize]) * static_cast<float>(mask[idx]) * scale + add[idx];
    }

};

template<typename Functor, typename T>
__global__ void FusedBiasMaskScaleAddKernel(Functor functor, T *x, T *y) {
    unsigned int global_tid = blockDim.x * blockIdx.x + threadIdx.x;

    // In case number of threads < total size of data
    for (auto i = global_tid; i < N; i += gridDim.x * blockDim.x) {
        y[i] = functor.compute(x, i);
    }
}

void FusedBiasMaskScaleAdd_CPU(const float *x, const float* bias, const int biasSize, const uint8_t *mask, const float scale, const float *add, float *y) {
    for (auto i = 0; i < N; i++) {
        y[i] = (x[i] + bias[i % biasSize]) * static_cast<float>(mask[i]) * scale + add[i];
    }
}

bool checkResults(const float *res_cpu, const float *res_gpu) {
    for (unsigned int i = 0 ; i < N; i++) {
        if (res_cpu[i] != res_gpu[i]) {
            std::cout << "Check Failed: res=" << res_gpu[i] << " Ground Truth=" << res_cpu[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    float *hx, *hy, *hy_cpu;
    float *dx, *dy;

    float scale = 0.5f;
    float biasSize = 10; // recurrently apply 10 bias to all x
    uint8_t *h_mask, *d_mask;
    float *h_bias, *d_bias;
    float *h_add, *d_add;

    // Allocate CPU memory
    hx = (float*)malloc(N * sizeof(float));
    hy = (float*)malloc(N * sizeof(float));
    hy_cpu = (float*)malloc(N * sizeof(float));
    h_bias = (float*)malloc(biasSize * sizeof(float));
    h_mask = (uint8_t*)malloc(N * sizeof(uint8_t));
    h_add = (float*)malloc(N * sizeof(float));

    // Init data
    for (auto i = 0; i < biasSize; i++) {
        h_bias[i] = static_cast<float>(i);
    }
    for (auto i = 0; i < N; i++) {
        hx[i] = static_cast<float>(i);
        h_mask[i] = static_cast<uint8_t>(i % 2);  // 010101...
        h_add[i] = static_cast<float>(i);
    }


    // Allocate GPU memory
    hipMalloc((void**)&dx, N * sizeof(float));
    hipMalloc((void**)&d_bias, biasSize * sizeof(float));
    hipMalloc((void**)&d_mask, N * sizeof(uint8_t));
    hipMalloc((void**)&d_add, N * sizeof(float));
    hipMalloc((void**)&dy, N * sizeof(float));

    // Convert float input to half input
    hipMemcpy(dx, hx, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, h_bias, biasSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, N * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_add, h_add, N * sizeof(float), hipMemcpyHostToDevice);

    // Get device property and calculate block needed
    hipDeviceProp_t deviceProp{};
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);
    const int blockSize = 1024;  // max threads per block
    const int gridSize = (N + blockSize - 1) / blockSize;

    // Call kernel function
    float millisecond = 0.0f;
    dim3 Grid(gridSize);  // number of blocks
    dim3 Block(blockSize);  // number of threads
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    auto fused_func = BiasMaskScaleAddFunctor<float>(d_bias, biasSize, d_mask, scale, d_add);
    FusedBiasMaskScaleAddKernel<BiasMaskScaleAddFunctor<float>, float><<<Grid, Block>>>(fused_func, dx, dy);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millisecond, start, stop);


    // Copy GPU result to CPU
    hipMemcpy(hy, dy, N * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Time Spent: " << millisecond << "ms." << std::endl;

    // Check results
    FusedBiasMaskScaleAdd_CPU(hx, h_bias, biasSize, h_mask, scale, h_add, hy_cpu);
    checkResults(hy_cpu, hy);


    // Free resources
    hipFree(dx);
    hipFree(d_bias);
    hipFree(d_mask);
    hipFree(d_add);
    hipFree(dy);

    free(hx);
    free(hy);
    free(hy_cpu);
    free(h_bias);
    free(h_mask);
    free(h_add);

    return 0;
}
