#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


//v4: 最后一个warp不用参与__syncthreads
//latency: 0.694ms
__device__ void WarpSharedMemReduce(volatile float* smem, int tid){
    // CUDA不保证所有的shared memory读操作都能在写操作之前完成，因此存在竞争关系，可能导致结果错误
    // 比如smem[tid] += smem[tid + 16] => smem[0] += smem[16], smem[16] += smem[32]
    // 此时L9中smem[16]的读和写到底谁在前谁在后，这是不确定的，所以在Volta架构后最后加入中间寄存器(L11)配合syncwarp和volatile(使得不会看见其他线程更新smem上的结果)保证读写依赖
    float x = smem[tid];
    if (blockDim.x >= 64) {
      x += smem[tid + 32]; __syncwarp();
      smem[tid] = x; __syncwarp();
    }
    x += smem[tid + 16]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 8]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 4]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 2]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 1]; __syncwarp();
    smem[tid] = x; __syncwarp();
}
// Note: using blockSize as a template arg can benefit from NVCC compiler optimization, 
// which is better than using blockDim.x that is known in runtime.
template<int blockSize>
__global__ void reduce_v4(float *d_in,float *d_out){
    __shared__ float smem[blockSize];
    // 泛指当前线程在其block内的id
    int tid = threadIdx.x;
    // 泛指当前线程在所有block范围内的全局id, *2代表当前block要处理2*blocksize的数据
    // ep. blocksize = 2, blockIdx.x = 1, when tid = 0, gtid = 4, gtid + blockSize = 6; when tid = 1, gtid = 5, gtid + blockSize = 7
    // ep. blocksize = 2, blockIdx.x = 0, when tid = 0, gtid = 0, gtid + blockSize = 2; when tid = 1, gtid = 1, gtid + blockSize = 3
    // so, we can understand L38, one thread handle data located in tid and tid + blockSize 
    int i = blockIdx.x * (blockSize * 2) + threadIdx.x;
    // load: 每个线程加载两个元素到shared mem对应位置
    smem[tid] = d_in[i] + d_in[i + blockSize];
    __syncthreads();

    // 基于v3改进：把最后一个warp抽离出来reduce，避免多做一次sync threads
    // 此时一个block对d_in这块数据的reduce sum结果保存在id为0的线程上面
    for (int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }

    // last warp拎出来单独作reduce
    if (tid < 32) {
        WarpSharedMemReduce(smem, tid);
    }
    // store: 哪里来回哪里去，把reduce结果写回显存
    // GridSize个block内部的reduce sum已得出，保存到d_out的每个索引位置
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    //printf("%f", res);
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    //const int N = 32 * 1024 * 1024;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize / 2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v4<blockSize / 2><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d \n", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < GridSize;i++){
            printf("resPerBlock : %lf ",out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v4 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}
